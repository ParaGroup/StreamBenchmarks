
#include <hip/hip_runtime.h>
 extern "C"
__global__ void filter_kernel(float *v1, float *v2, float *v3, float *v4, int *flags, int size)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; // id of the thread in the kernel
    int num_threads = gridDim.x * blockDim.x; // number of threads in the kernel
    for (int i=id; i<size; i+=num_threads) {
        if (v1[i] < 0.9 && v2[i] < 0.9 && v3[i] < 0.9 && v4[i] < 0.9) {
            flags[i] = 1;
            v1[i] = (v1[i] + v2[i] + v3[i] + v4[i])/4;
            v2[i] = v3[i] = v4[i] = 0;
        }
        else {
            flags[i] = 0;
        }
    }
}
